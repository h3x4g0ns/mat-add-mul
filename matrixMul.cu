#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

__global__ void matMultKernel(float* A, float* B, float* C, int N) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  float sum = 0;
  if(row < N && col < N) {
    for(int i = 0; i < N; i++) {
      sum += A[row * N + i] * B[i * N + col];
    }
    C[row * N + col] = sum;
  }
}

void matMult(float* A, float* B, float* C, int N) {
  int size = N * N * sizeof(float);
  float *d_A, *d_B, *d_C;

  hipMalloc((void**) &d_A, size);
  hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
  hipMalloc((void**) &d_B, size);
  hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

  hipMalloc((void**) &d_C, size);

  dim3 threadsPerBlock(N, N);
  dim3 blocksPerGrid(1, 1);
  if (N*N > 1024){
    threadsPerBlock.x = 1024;
    threadsPerBlock.y = 1024;
    blocksPerGrid.x = ceil(double(N)/double(threadsPerBlock.x));
    blocksPerGrid.y = ceil(double(N)/double(threadsPerBlock.y));
  }

  matMultKernel<<<blocksPerGrid,threadsPerBlock>>>(d_A, d_B, d_C, N);

  hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

  hipFree(d_A); 
  hipFree(d_B); 
  hipFree(d_C);
}

int main() {
  int N = 2;
  float A[N*N], B[N*N], C[N*N];
  for(int i = 0; i < N*N; i++) {
    A[i] = i;
    B[i] = i;
  }
  matMult(A, B, C, N);
  for(int i = 0; i < N*N; i++) {
    printf("%f ", C[i]);
  }
  return 0;
}
